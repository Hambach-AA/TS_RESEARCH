#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <string>

__global__ void MuSigma(float* expansionTS_d, float* mu_d, float* sigma_d, int subseqLen, int more) {
    
    for (int i = 0; i < more; i++) {
        int index = (threadIdx.x + blockIdx.x * blockDim.x) + (i * gridDim.x * blockDim.x);
        float sum = 0;
        for (int j = index; j < index + subseqLen; j++) {
            sum += expansionTS_d[j];
        }
        mu_d[index] = sum / subseqLen;
        sum = 0;
        for (int j = index; j < index + subseqLen; j++) {
            sum += pow(expansionTS_d[j] - mu_d[index], 2);
        }
        sigma_d[index] = sqrtf(sum / subseqLen);
    }  
}

__global__ void Scalar(float* TS_d, float* scalar_d, int* shift_d, int* x_d, int subseqLen, int more, int scalarLen, int matrixLen) {

    for (int i = 0; i < more; i++) {
        int index_x = (threadIdx.x + blockIdx.x * blockDim.x) + (i * gridDim.x * blockDim.x);
        if (index_x < scalarLen) {
            index_x = x_d[(threadIdx.x + blockIdx.x * blockDim.x) + (i * gridDim.x * blockDim.x)];
            int x = index_x % matrixLen;
            int y = index_x / matrixLen;
            float sum = 0;
            for (int i = 0; i < subseqLen; i++) {
                sum += TS_d[x + i] * TS_d[y + i];
            }
            scalar_d[x + (y * matrixLen) - shift_d[y]] = sum;
        }
    }
}

extern "C" __declspec(dllexport) void cudaRan_MuSigma(float * ts, int sizeTS, int subseqLen, float* mu, float* sigma) {

    int sizeSubseq = sizeTS - subseqLen + 1;
    int block = 10;
    int thread = 100;
    int allTread = block * thread;
    int more = sizeSubseq / allTread + 1;
    int num = more * allTread;

    float *expansionTS = new float[num + subseqLen - 1] {} ;
    //float *mu = new float[num] {} ;
    //float *sigma = new float[num] {} ;
    
    for (int i = 0; i < sizeTS; i++) {
        expansionTS[i] = ts[i];
    }
    float* expansionTS_d;
    float* mu_d;
    float* sigma_d;

    hipMalloc((void**)&expansionTS_d, sizeof(float) * (num + subseqLen - 1));
    hipMalloc((void**)&mu_d, sizeof(float) * num);
    hipMalloc((void**)&sigma_d, sizeof(float) * num);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMemcpy(expansionTS_d, expansionTS, sizeof(float) * (num + subseqLen - 1), hipMemcpyKind::hipMemcpyHostToDevice);

    MuSigma <<<block, thread>>> (expansionTS_d, mu_d, sigma_d, subseqLen, more);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time work: %f\n", time);
    
    hipMemcpy(mu, mu_d, sizeof(float) * num, hipMemcpyKind::hipMemcpyDeviceToHost);
    hipMemcpy(sigma, sigma_d, sizeof(float) * num, hipMemcpyKind::hipMemcpyDeviceToHost);

    hipFree(expansionTS_d);
    hipFree(mu_d);
    hipFree(sigma_d);
}


extern "C" __declspec(dllexport) void cudaRan_Scalar(float* ts, int sizeTS, int subseqLen, float * scalar) {

    int sizeSubseq = sizeTS - subseqLen + 1;

    int block = 100;
    int thread = 100;
    int allTread = thread * block;

    int* shift = new int[sizeSubseq - 1] {1};
    for (int i = 2; i < sizeSubseq; i++) {
        shift[i - 1] = shift[i - 2] + i;
    }

    int more = shift[sizeSubseq - 2] / allTread + 1;

    int* buf = new int[sizeSubseq - 1] {0};
    for (int i = 1; i < sizeSubseq - 1; i++) {
        buf[i] = (sizeSubseq + 1) * i;
    }

    int* x = new int[shift[sizeSubseq - 2]];

    int io = 0;
    for (int i = sizeSubseq - 1; i > 0; i--) {
        for (int j = 0; j < i; j++) {
            x[io] = j + 1 + buf[sizeSubseq - 1 - i];
            io++;
        }
    }

    //float* scalar = new float[shift[sizeSubseq - 2]];
    
    float* TS_d;
    float* scalar_d;
    int* shift_d;
    int* x_d;

    hipMalloc((void**)&TS_d, sizeof(float) * sizeTS);
    hipMalloc((void**)&scalar_d, sizeof(float) * shift[sizeSubseq - 2]);
    hipMalloc((void**)&shift_d, sizeof(int) * (sizeSubseq - 1));
    hipMalloc((void**)&x_d, sizeof(int) * shift[sizeSubseq - 2]);

    float time;

    hipMemcpy(TS_d, ts, sizeof(float) * sizeTS, hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(shift_d, shift, sizeof(int) * (sizeSubseq - 1), hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(x_d, x, sizeof(int) * shift[sizeSubseq - 2], hipMemcpyKind::hipMemcpyHostToDevice);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    Scalar <<<block, thread >>> (TS_d, scalar_d, shift_d, x_d, subseqLen, more, shift[sizeSubseq - 2], sizeSubseq);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time work: %f\n", time);

    hipMemcpy(scalar, scalar_d, sizeof(float) * shift[sizeSubseq - 2], hipMemcpyKind::hipMemcpyDeviceToHost);

    hipFree(TS_d);
    hipFree(scalar_d);
    hipFree(shift_d);
    hipFree(x_d);
}
